#include "hip/hip_runtime.h"
#include "bindings.h"
#include "helpers.cuh"
#include "third_party/glm/glm/glm.hpp"
#include "third_party/glm/glm/gtc/type_ptr.hpp"
#include "utils.cuh"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace cg = cooperative_groups;

/****************************************************************************
 * Quat-Scale to Covariance and Precision
 ****************************************************************************/

__global__ void
quat_scale_to_covar_preci_fwd_kernel(const uint32_t N,
                                     const float *__restrict__ quats,  // [N, 4]
                                     const float *__restrict__ scales, // [N, 3]
                                     const bool triu,
                                     // outputs
                                     float *__restrict__ covars, // [N, 3, 3] or [N, 6]
                                     float *__restrict__ precis  // [N, 3, 3] or [N, 6]
) {
    // parallelize over N.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= N) {
        return;
    }

    // shift pointers to the current gaussian
    quats += idx * 4;
    scales += idx * 3;

    // compute the matrices
    glm::mat3 covar, preci;
    quat_scale_to_covar_preci(glm::make_vec4(quats), glm::make_vec3(scales),
                              covars ? &covar : nullptr, precis ? &preci : nullptr);

    // write to outputs: glm is column-major but we want row-major
    if (covars != nullptr) {
        if (triu) {
            covars += idx * 6;
            covars[0] = covar[0][0];
            covars[1] = covar[0][1];
            covars[2] = covar[0][2];
            covars[3] = covar[1][1];
            covars[4] = covar[1][2];
            covars[5] = covar[2][2];
        } else {
            covars += idx * 9;
            PRAGMA_UNROLL
            for (uint32_t i = 0; i < 3; i++) { // rows
                PRAGMA_UNROLL
                for (uint32_t j = 0; j < 3; j++) { // cols
                    covars[i * 3 + j] = covar[j][i];
                }
            }
        }
    }
    if (precis != nullptr) {
        if (triu) {
            precis += idx * 6;
            precis[0] = preci[0][0];
            precis[1] = preci[0][1];
            precis[2] = preci[0][2];
            precis[3] = preci[1][1];
            precis[4] = preci[1][2];
            precis[5] = preci[2][2];
        } else {
            precis += idx * 9;
            PRAGMA_UNROLL
            for (uint32_t i = 0; i < 3; i++) { // rows
                PRAGMA_UNROLL
                for (uint32_t j = 0; j < 3; j++) { // cols
                    precis[i * 3 + j] = preci[j][i];
                }
            }
        }
    }
}

__global__ void quat_scale_to_covar_preci_bwd_kernel(
    const uint32_t N,
    // fwd inputs
    const float *__restrict__ quats,  // [N, 4]
    const float *__restrict__ scales, // [N, 3]
    // grad outputs
    const float *__restrict__ v_covars, // [N, 3, 3] or [N, 6]
    const float *__restrict__ v_precis, // [N, 3, 3] or [N, 6]
    const bool triu,
    // grad inputs
    float *__restrict__ v_scales, // [N, 3]
    float *__restrict__ v_quats   // [N, 4]
) {
    // parallelize over N.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= N) {
        return;
    }

    // shift pointers to the current gaussian
    v_scales += idx * 3;
    v_quats += idx * 4;

    glm::vec4 quat = glm::make_vec4(quats + idx * 4);
    glm::vec3 scale = glm::make_vec3(scales + idx * 3);
    glm::mat3 rotmat = quat_to_rotmat(quat);

    glm::vec4 v_quat(0.f);
    glm::vec3 v_scale(0.f);
    if (v_covars != nullptr) {
        // glm is column-major, input is row-major
        glm::mat3 v_covar;
        if (triu) {
            v_covars += idx * 6;
            v_covar = glm::mat3(v_covars[0], v_covars[1] * .5f, v_covars[2] * .5f,
                                v_covars[1] * .5f, v_covars[3], v_covars[4] * .5f,
                                v_covars[2] * .5f, v_covars[4] * .5f, v_covars[5]);
        } else {
            v_covars += idx * 9;
            v_covar = glm::transpose(glm::make_mat3(v_covars));
        }
        quat_scale_to_covar_vjp(quat, scale, rotmat, v_covar, v_quat, v_scale);
    }
    if (v_precis != nullptr) {
        // glm is column-major, input is row-major
        glm::mat3 v_preci;
        if (triu) {
            v_precis += idx * 6;
            v_preci = glm::mat3(v_precis[0], v_precis[1] * .5f, v_precis[2] * .5f,
                                v_precis[1] * .5f, v_precis[3], v_precis[4] * .5f,
                                v_precis[2] * .5f, v_precis[4] * .5f, v_precis[5]);
        } else {
            v_precis += idx * 9;
            v_preci = glm::transpose(glm::make_mat3(v_precis));
        }
        quat_scale_to_preci_vjp(quat, scale, rotmat, v_preci, v_quat, v_scale);
    }

    // write out results
    PRAGMA_UNROLL
    for (uint32_t k = 0; k < 3; ++k) {
        v_scales[k] = v_scale[k];
    }
    PRAGMA_UNROLL
    for (uint32_t k = 0; k < 4; ++k) {
        v_quats[k] = v_quat[k];
    }
}

std::tuple<torch::Tensor, torch::Tensor>
quat_scale_to_covar_preci_fwd_tensor(const torch::Tensor &quats,  // [N, 4]
                                     const torch::Tensor &scales, // [N, 3]
                                     const bool compute_covar, const bool compute_preci,
                                     const bool triu) {
    DEVICE_GUARD(quats);
    CHECK_INPUT(quats);
    CHECK_INPUT(scales);

    uint32_t N = quats.size(0);

    torch::Tensor covars, precis;
    if (compute_covar) {
        if (triu) {
            covars = torch::empty({N, 6}, quats.options());
        } else {
            covars = torch::empty({N, 3, 3}, quats.options());
        }
    }
    if (compute_preci) {
        if (triu) {
            precis = torch::empty({N, 6}, quats.options());
        } else {
            precis = torch::empty({N, 3, 3}, quats.options());
        }
    }

    if (N) {
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
        quat_scale_to_covar_preci_fwd_kernel<<<(N + N_THREADS - 1) / N_THREADS,
                                               N_THREADS, 0, stream>>>(
            N, quats.data_ptr<float>(), scales.data_ptr<float>(), triu,
            compute_covar ? covars.data_ptr<float>() : nullptr,
            compute_preci ? precis.data_ptr<float>() : nullptr);
    }
    return std::make_tuple(covars, precis);
}

std::tuple<torch::Tensor, torch::Tensor> quat_scale_to_covar_preci_bwd_tensor(
    const torch::Tensor &quats,                  // [N, 4]
    const torch::Tensor &scales,                 // [N, 3]
    const at::optional<torch::Tensor> &v_covars, // [N, 3, 3] or [N, 6]
    const at::optional<torch::Tensor> &v_precis, // [N, 3, 3] or [N, 6]
    const bool triu) {
    DEVICE_GUARD(quats);
    CHECK_INPUT(quats);
    CHECK_INPUT(scales);
    if (v_covars.has_value()) {
        CHECK_INPUT(v_covars.value());
    }
    if (v_precis.has_value()) {
        CHECK_INPUT(v_precis.value());
    }

    uint32_t N = quats.size(0);

    torch::Tensor v_scales = torch::empty_like(scales);
    torch::Tensor v_quats = torch::empty_like(quats);

    if (N) {
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
        quat_scale_to_covar_preci_bwd_kernel<<<(N + N_THREADS - 1) / N_THREADS,
                                               N_THREADS, 0, stream>>>(
            N, quats.data_ptr<float>(), scales.data_ptr<float>(),
            v_covars.has_value() ? v_covars.value().data_ptr<float>() : nullptr,
            v_precis.has_value() ? v_precis.value().data_ptr<float>() : nullptr, triu,
            v_scales.data_ptr<float>(), v_quats.data_ptr<float>());
    }

    return std::make_tuple(v_quats, v_scales);
}

/****************************************************************************
 * Perspective Projection
 ****************************************************************************/

__global__ void persp_proj_fwd_kernel(const uint32_t C, const uint32_t N,
                                      const float *__restrict__ means,  // [C, N, 3]
                                      const float *__restrict__ covars, // [C, N, 3, 3]
                                      const float *__restrict__ Ks,     // [C, 3, 3]
                                      const uint32_t width, const uint32_t height,
                                      float *__restrict__ means2d, // [C, N, 2]
                                      float *__restrict__ covars2d // [C, N, 2, 2]
) { // parallelize over C * N.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= C * N) {
        return;
    }
    const uint32_t cid = idx / N; // camera id
    // const uint32_t gid = idx % N; // gaussian id

    // shift pointers to the current camera and gaussian
    means += idx * 3;
    covars += idx * 9;
    Ks += cid * 9;
    means2d += idx * 2;
    covars2d += idx * 4;

    float fx = Ks[0], cx = Ks[2], fy = Ks[4], cy = Ks[5];
    glm::mat2 covar2d;
    glm::vec2 mean2d;
    persp_proj(glm::make_vec3(means), glm::make_mat3(covars), fx, fy, cx, cy, width,
               height, covar2d, mean2d);

    // write to outputs: glm is column-major but we want row-major
    PRAGMA_UNROLL
    for (uint32_t i = 0; i < 2; i++) { // rows
        PRAGMA_UNROLL
        for (uint32_t j = 0; j < 2; j++) { // cols
            covars2d[i * 2 + j] = covar2d[j][i];
        }
    }
    PRAGMA_UNROLL
    for (uint32_t i = 0; i < 2; i++) {
        means2d[i] = mean2d[i];
    }
}

__global__ void
persp_proj_bwd_kernel(const uint32_t C, const uint32_t N,
                      const float *__restrict__ means,  // [C, N, 3]
                      const float *__restrict__ covars, // [C, N, 3, 3]
                      const float *__restrict__ Ks,     // [C, 3, 3]
                      const uint32_t width, const uint32_t height,
                      const float *__restrict__ v_means2d,  // [C, N, 2]
                      const float *__restrict__ v_covars2d, // [C, N, 2, 2]
                      float *__restrict__ v_means,          // [C, N, 3]
                      float *__restrict__ v_covars          // [C, N, 3, 3]
) {
    // parallelize over C * N.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= C * N) {
        return;
    }
    const uint32_t cid = idx / N; // camera id
    // const uint32_t gid = idx % N; // gaussian id

    // shift pointers to the current camera and gaussian
    means += idx * 3;
    covars += idx * 9;
    v_means += idx * 3;
    v_covars += idx * 9;
    Ks += cid * 9;
    v_means2d += idx * 2;
    v_covars2d += idx * 4;

    float fx = Ks[0], cx = Ks[2], fy = Ks[4], cy = Ks[5];
    glm::mat3 v_covar(0.f);
    glm::vec3 v_mean(0.f);
    persp_proj_vjp(glm::make_vec3(means), glm::make_mat3(covars), fx, fy, cx, cy, width,
                   height, glm::transpose(glm::make_mat2(v_covars2d)),
                   glm::make_vec2(v_means2d), v_mean, v_covar);

    // write to outputs: glm is column-major but we want row-major
    PRAGMA_UNROLL
    for (uint32_t i = 0; i < 3; i++) { // rows
        PRAGMA_UNROLL
        for (uint32_t j = 0; j < 3; j++) { // cols
            v_covars[i * 3 + j] = v_covar[j][i];
        }
    }

    PRAGMA_UNROLL
    for (uint32_t i = 0; i < 3; i++) {
        v_means[i] = v_mean[i];
    }
}

std::tuple<torch::Tensor, torch::Tensor>
persp_proj_fwd_tensor(const torch::Tensor &means,  // [C, N, 3]
                      const torch::Tensor &covars, // [C, N, 3, 3]
                      const torch::Tensor &Ks,     // [C, 3, 3]
                      const uint32_t width, const uint32_t height) {
    DEVICE_GUARD(means);
    CHECK_INPUT(means);
    CHECK_INPUT(covars);
    CHECK_INPUT(Ks);

    uint32_t C = means.size(0);
    uint32_t N = means.size(1);

    torch::Tensor means2d = torch::empty({C, N, 2}, means.options());
    torch::Tensor covars2d = torch::empty({C, N, 2, 2}, covars.options());

    if (C && N) {
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
        persp_proj_fwd_kernel<<<(C * N + N_THREADS - 1) / N_THREADS, N_THREADS, 0,
                                stream>>>(
            C, N, means.data_ptr<float>(), covars.data_ptr<float>(),
            Ks.data_ptr<float>(), width, height, means2d.data_ptr<float>(),
            covars2d.data_ptr<float>());
    }
    return std::make_tuple(means2d, covars2d);
}

std::tuple<torch::Tensor, torch::Tensor>
persp_proj_bwd_tensor(const torch::Tensor &means,  // [C, N, 3]
                      const torch::Tensor &covars, // [C, N, 3, 3]
                      const torch::Tensor &Ks,     // [C, 3, 3]
                      const uint32_t width, const uint32_t height,
                      const torch::Tensor &v_means2d, // [C, N, 2]
                      const torch::Tensor &v_covars2d // [C, N, 2, 2]
) {
    DEVICE_GUARD(means);
    CHECK_INPUT(means);
    CHECK_INPUT(covars);
    CHECK_INPUT(Ks);
    CHECK_INPUT(v_means2d);
    CHECK_INPUT(v_covars2d);

    uint32_t C = means.size(0);
    uint32_t N = means.size(1);

    torch::Tensor v_means = torch::empty({C, N, 3}, means.options());
    torch::Tensor v_covars = torch::empty({C, N, 3, 3}, means.options());

    if (C && N) {
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
        persp_proj_bwd_kernel<<<(C * N + N_THREADS - 1) / N_THREADS, N_THREADS, 0,
                                stream>>>(
            C, N, means.data_ptr<float>(), covars.data_ptr<float>(),
            Ks.data_ptr<float>(), width, height, v_means2d.data_ptr<float>(),
            v_covars2d.data_ptr<float>(), v_means.data_ptr<float>(),
            v_covars.data_ptr<float>());
    }
    return std::make_tuple(v_means, v_covars);
}

/****************************************************************************
 * World to Camera Transformation
 ****************************************************************************/

__global__ void world_to_cam_fwd_kernel(const uint32_t C, const uint32_t N,
                                        const float *__restrict__ means,    // [N, 3]
                                        const float *__restrict__ covars,   // [N, 3, 3]
                                        const float *__restrict__ viewmats, // [C, 4, 4]
                                        float *__restrict__ means_c,        // [C, N, 3]
                                        float *__restrict__ covars_c // [C, N, 3, 3]
) { // parallelize over C * N.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= C * N) {
        return;
    }
    const uint32_t cid = idx / N; // camera id
    const uint32_t gid = idx % N; // gaussian id

    // shift pointers to the current camera and gaussian
    means += gid * 3;
    covars += gid * 9;
    viewmats += cid * 16;

    // glm is column-major but input is row-major
    glm::mat3 R = glm::mat3(viewmats[0], viewmats[4], viewmats[8], // 1st column
                            viewmats[1], viewmats[5], viewmats[9], // 2nd column
                            viewmats[2], viewmats[6], viewmats[10] // 3rd column
    );
    glm::vec3 t = glm::vec3(viewmats[3], viewmats[7], viewmats[11]);

    if (means_c != nullptr) {
        glm::vec3 mean_c;
        pos_world_to_cam(R, t, glm::make_vec3(means), mean_c);
        means_c += idx * 3;
        PRAGMA_UNROLL
        for (uint32_t i = 0; i < 3; i++) { // rows
            means_c[i] = mean_c[i];
        }
    }

    // write to outputs: glm is column-major but we want row-major
    if (covars_c != nullptr) {
        glm::mat3 covar_c;
        covar_world_to_cam(R, glm::make_mat3(covars), covar_c);
        covars_c += idx * 9;
        PRAGMA_UNROLL
        for (uint32_t i = 0; i < 3; i++) { // rows
            PRAGMA_UNROLL
            for (uint32_t j = 0; j < 3; j++) { // cols
                covars_c[i * 3 + j] = covar_c[j][i];
            }
        }
    }
}

__global__ void
world_to_cam_bwd_kernel(const uint32_t C, const uint32_t N,
                        const float *__restrict__ means,      // [N, 3]
                        const float *__restrict__ covars,     // [N, 3, 3]
                        const float *__restrict__ viewmats,   // [C, 4, 4]
                        const float *__restrict__ v_means_c,  // [C, N, 3]
                        const float *__restrict__ v_covars_c, // [C, N, 3, 3]
                        float *__restrict__ v_means,          // [N, 3]
                        float *__restrict__ v_covars,         // [N, 3, 3]
                        float *__restrict__ v_viewmats        // [C, 4, 4]
) {
    // parallelize over C * N.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= C * N) {
        return;
    }
    const uint32_t cid = idx / N; // camera id
    const uint32_t gid = idx % N; // gaussian id

    // shift pointers to the current camera and gaussian
    means += gid * 3;
    covars += gid * 9;
    viewmats += cid * 16;

    // glm is column-major but input is row-major
    glm::mat3 R = glm::mat3(viewmats[0], viewmats[4], viewmats[8], // 1st column
                            viewmats[1], viewmats[5], viewmats[9], // 2nd column
                            viewmats[2], viewmats[6], viewmats[10] // 3rd column
    );
    glm::vec3 t = glm::vec3(viewmats[3], viewmats[7], viewmats[11]);

    glm::vec3 v_mean(0.f);
    glm::mat3 v_covar(0.f);
    glm::mat3 v_R(0.f);
    glm::vec3 v_t(0.f);

    if (v_means_c != nullptr) {
        glm::vec3 v_mean_c = glm::make_vec3(v_means_c + idx * 3);
        pos_world_to_cam_vjp(R, t, glm::make_vec3(means), v_mean_c, v_R, v_t, v_mean);
    }
    if (v_covars_c != nullptr) {
        glm::mat3 v_covar_c = glm::transpose(glm::make_mat3(v_covars_c + idx * 9));
        covar_world_to_cam_vjp(R, glm::make_mat3(covars), v_covar_c, v_R, v_covar);
    }

    // #if __CUDA_ARCH__ >= 700
    // write out results with warp-level reduction
    auto warp = cg::tiled_partition<32>(cg::this_thread_block());
    auto warp_group_g = cg::labeled_partition(warp, gid);
    if (v_means != nullptr) {
        warpSum(v_mean, warp_group_g);
        if (warp_group_g.thread_rank() == 0) {
            v_means += gid * 3;
            PRAGMA_UNROLL
            for (uint32_t i = 0; i < 3; i++) {
                atomicAdd(v_means + i, v_mean[i]);
            }
        }
    }
    if (v_covars != nullptr) {
        warpSum(v_covar, warp_group_g);
        if (warp_group_g.thread_rank() == 0) {
            v_covars += gid * 9;
            PRAGMA_UNROLL
            for (uint32_t i = 0; i < 3; i++) { // rows
                PRAGMA_UNROLL
                for (uint32_t j = 0; j < 3; j++) { // cols
                    atomicAdd(v_covars + i * 3 + j, v_covar[j][i]);
                }
            }
        }
    }
    if (v_viewmats != nullptr) {
        auto warp_group_c = cg::labeled_partition(warp, cid);
        warpSum(v_R, warp_group_c);
        warpSum(v_t, warp_group_c);
        if (warp_group_c.thread_rank() == 0) {
            v_viewmats += cid * 16;
            PRAGMA_UNROLL
            for (uint32_t i = 0; i < 3; i++) { // rows
                PRAGMA_UNROLL
                for (uint32_t j = 0; j < 3; j++) { // cols
                    atomicAdd(v_viewmats + i * 4 + j, v_R[j][i]);
                }
                atomicAdd(v_viewmats + i * 4 + 3, v_t[i]);
            }
        }
    }
}

std::tuple<torch::Tensor, torch::Tensor>
world_to_cam_fwd_tensor(const torch::Tensor &means,   // [N, 3]
                        const torch::Tensor &covars,  // [N, 3, 3]
                        const torch::Tensor &viewmats // [C, 4, 4]
) {
    DEVICE_GUARD(means);
    CHECK_INPUT(means);
    CHECK_INPUT(covars);
    CHECK_INPUT(viewmats);

    uint32_t N = means.size(0);
    uint32_t C = viewmats.size(0);

    torch::Tensor means_c = torch::empty({C, N, 3}, means.options());
    torch::Tensor covars_c = torch::empty({C, N, 3, 3}, means.options());

    if (C && N) {
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
        world_to_cam_fwd_kernel<<<(C * N + N_THREADS - 1) / N_THREADS, N_THREADS, 0,
                                  stream>>>(
            C, N, means.data_ptr<float>(), covars.data_ptr<float>(),
            viewmats.data_ptr<float>(), means_c.data_ptr<float>(),
            covars_c.data_ptr<float>());
    }
    return std::make_tuple(means_c, covars_c);
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor>
world_to_cam_bwd_tensor(const torch::Tensor &means,                    // [N, 3]
                        const torch::Tensor &covars,                   // [N, 3, 3]
                        const torch::Tensor &viewmats,                 // [C, 4, 4]
                        const at::optional<torch::Tensor> &v_means_c,  // [C, N, 3]
                        const at::optional<torch::Tensor> &v_covars_c, // [C, N, 3, 3]
                        const bool means_requires_grad, const bool covars_requires_grad,
                        const bool viewmats_requires_grad) {
    DEVICE_GUARD(means);
    CHECK_INPUT(means);
    CHECK_INPUT(covars);
    CHECK_INPUT(viewmats);
    if (v_means_c.has_value()) {
        CHECK_INPUT(v_means_c.value());
    }
    if (v_covars_c.has_value()) {
        CHECK_INPUT(v_covars_c.value());
    }
    uint32_t N = means.size(0);
    uint32_t C = viewmats.size(0);

    torch::Tensor v_means, v_covars, v_viewmats;
    if (means_requires_grad) {
        v_means = torch::zeros({N, 3}, means.options());
    }
    if (covars_requires_grad) {
        v_covars = torch::zeros({N, 3, 3}, means.options());
    }
    if (viewmats_requires_grad) {
        v_viewmats = torch::zeros({C, 4, 4}, means.options());
    }

    if (C && N) {
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
        world_to_cam_bwd_kernel<<<(C * N + N_THREADS - 1) / N_THREADS, N_THREADS, 0,
                                  stream>>>(
            C, N, means.data_ptr<float>(), covars.data_ptr<float>(),
            viewmats.data_ptr<float>(),
            v_means_c.has_value() ? v_means_c.value().data_ptr<float>() : nullptr,
            v_covars_c.has_value() ? v_covars_c.value().data_ptr<float>() : nullptr,
            means_requires_grad ? v_means.data_ptr<float>() : nullptr,
            covars_requires_grad ? v_covars.data_ptr<float>() : nullptr,
            viewmats_requires_grad ? v_viewmats.data_ptr<float>() : nullptr);
    }
    return std::make_tuple(v_means, v_covars, v_viewmats);
}

/****************************************************************************
 * Projection of Gaussians
 ****************************************************************************/

__global__ void
fully_fused_projection_fwd_kernel(const uint32_t C, const uint32_t N,
                                  const float *__restrict__ means,    // [N, 3]
                                  const float *__restrict__ covars,   // [N, 6] optional
                                  const float *__restrict__ quats,    // [N, 4] optional
                                  const float *__restrict__ scales,   // [N, 3] optional
                                  const float *__restrict__ viewmats, // [C, 4, 4]
                                  const float *__restrict__ Ks,       // [C, 3, 3]
                                  const int32_t image_width, const int32_t image_height,
                                  const float eps2d, const float near_plane,
                                  const float far_plane, const float radius_clip,
                                  // outputs
                                  int32_t *__restrict__ radii,      // [C, N]
                                  float *__restrict__ means2d,      // [C, N, 2]
                                  float *__restrict__ depths,       // [C, N]
                                  float *__restrict__ conics,       // [C, N, 3]
                                  float *__restrict__ compensations // [C, N] optional
) {
    // parallelize over C * N.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= C * N) {
        return;
    }
    const uint32_t cid = idx / N; // camera id
    const uint32_t gid = idx % N; // gaussian id

    // shift pointers to the current camera and gaussian
    means += gid * 3;
    viewmats += cid * 16;
    Ks += cid * 9;

    // glm is column-major but input is row-major
    glm::mat3 R = glm::mat3(viewmats[0], viewmats[4], viewmats[8], // 1st column
                            viewmats[1], viewmats[5], viewmats[9], // 2nd column
                            viewmats[2], viewmats[6], viewmats[10] // 3rd column
    );
    glm::vec3 t = glm::vec3(viewmats[3], viewmats[7], viewmats[11]);

    // transform Gaussian center to camera space
    glm::vec3 mean_c;
    pos_world_to_cam(R, t, glm::make_vec3(means), mean_c);
    if (mean_c.z < near_plane || mean_c.z > far_plane) {
        radii[idx] = 0;
        return;
    }

    // transform Gaussian covariance to camera space
    glm::mat3 covar;
    if (covars != nullptr) {
        covars += gid * 6;
        covar = glm::mat3(covars[0], covars[1], covars[2], // 1st column
                          covars[1], covars[3], covars[4], // 2nd column
                          covars[2], covars[4], covars[5]  // 3rd column
        );
    } else {
        // compute from quaternions and scales
        quats += gid * 4;
        scales += gid * 3;
        quat_scale_to_covar_preci(glm::make_vec4(quats), glm::make_vec3(scales), &covar,
                                  nullptr);
    }
    glm::mat3 covar_c;
    covar_world_to_cam(R, covar, covar_c);

    // perspective projection
    glm::mat2 covar2d;
    glm::vec2 mean2d;
    persp_proj(mean_c, covar_c, Ks[0], Ks[4], Ks[2], Ks[5], image_width, image_height,
               covar2d, mean2d);

    float compensation;
    float det = add_blur(eps2d, covar2d, compensation);
    if (det <= 0.f) {
        radii[idx] = 0;
        return;
    }

    // compute the inverse of the 2d covariance
    glm::mat2 covar2d_inv;
    inverse(covar2d, covar2d_inv);

    // take 3 sigma as the radius (non differentiable)
    float b = 0.5f * (covar2d[0][0] + covar2d[1][1]);
    float v1 = b + sqrt(max(0.01f, b * b - det));
    float radius = ceil(3.f * sqrt(v1));
    // float v2 = b - sqrt(max(0.1f, b * b - det));
    // float radius = ceil(3.f * sqrt(max(v1, v2)));

    if (radius <= radius_clip) {
        radii[idx] = 0;
        return;
    }

    // mask out gaussians outside the image region
    if (mean2d.x + radius <= 0 || mean2d.x - radius >= image_width ||
        mean2d.y + radius <= 0 || mean2d.y - radius >= image_height) {
        radii[idx] = 0;
        return;
    }

    // write to outputs
    radii[idx] = (int32_t)radius;
    means2d[idx * 2] = mean2d.x;
    means2d[idx * 2 + 1] = mean2d.y;
    depths[idx] = mean_c.z;
    conics[idx * 3] = covar2d_inv[0][0];
    conics[idx * 3 + 1] = covar2d_inv[0][1];
    conics[idx * 3 + 2] = covar2d_inv[1][1];
    if (compensations != nullptr) {
        compensations[idx] = compensation;
    }
}

__global__ void fully_fused_projection_bwd_kernel(
    // fwd inputs
    const uint32_t C, const uint32_t N,
    const float *__restrict__ means,    // [N, 3]
    const float *__restrict__ covars,   // [N, 6] optional
    const float *__restrict__ quats,    // [N, 4] optional
    const float *__restrict__ scales,   // [N, 3] optional
    const float *__restrict__ viewmats, // [C, 4, 4]
    const float *__restrict__ Ks,       // [C, 3, 3]
    const int32_t image_width, const int32_t image_height, const float eps2d,
    // fwd outputs
    const int32_t *__restrict__ radii,       // [C, N]
    const float *__restrict__ conics,        // [C, N, 3]
    const float *__restrict__ compensations, // [C, N] optional
    // grad outputs
    const float *__restrict__ v_means2d,       // [C, N, 2]
    const float *__restrict__ v_depths,        // [C, N]
    const float *__restrict__ v_conics,        // [C, N, 3]
    const float *__restrict__ v_compensations, // [C, N] optional
    // grad inputs
    float *__restrict__ v_means,   // [N, 3]
    float *__restrict__ v_covars,  // [N, 6] optional
    float *__restrict__ v_quats,   // [N, 4] optional
    float *__restrict__ v_scales,  // [N, 3] optional
    float *__restrict__ v_viewmats // [C, 4, 4] optional
) {
    // parallelize over C * N.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= C * N || radii[idx] <= 0) {
        return;
    }
    const uint32_t cid = idx / N; // camera id
    const uint32_t gid = idx % N; // gaussian id

    // shift pointers to the current camera and gaussian
    means += gid * 3;
    viewmats += cid * 16;
    Ks += cid * 9;

    conics += idx * 3;

    v_means2d += idx * 2;
    v_depths += idx;
    v_conics += idx * 3;

    // vjp: compute the inverse of the 2d covariance
    glm::mat2 covar2d_inv = glm::mat2(conics[0], conics[1], conics[1], conics[2]);
    glm::mat2 v_covar2d_inv =
        glm::mat2(v_conics[0], v_conics[1] * .5f, v_conics[1] * .5f, v_conics[2]);
    glm::mat2 v_covar2d(0.f);
    inverse_vjp(covar2d_inv, v_covar2d_inv, v_covar2d);

    if (v_compensations != nullptr) {
        // vjp: compensation term
        const float compensation = compensations[idx];
        const float v_compensation = v_compensations[idx];
        add_blur_vjp(eps2d, covar2d_inv, compensation, v_compensation, v_covar2d);
    }

    // transform Gaussian to camera space
    glm::mat3 R = glm::mat3(viewmats[0], viewmats[4], viewmats[8], // 1st column
                            viewmats[1], viewmats[5], viewmats[9], // 2nd column
                            viewmats[2], viewmats[6], viewmats[10] // 3rd column
    );
    glm::vec3 t = glm::vec3(viewmats[3], viewmats[7], viewmats[11]);

    glm::mat3 covar;
    glm::vec4 quat;
    glm::vec3 scale;
    if (covars != nullptr) {
        covars += gid * 6;
        covar = glm::mat3(covars[0], covars[1], covars[2], // 1st column
                          covars[1], covars[3], covars[4], // 2nd column
                          covars[2], covars[4], covars[5]  // 3rd column
        );
    } else {
        // compute from quaternions and scales
        quat = glm::make_vec4(quats + gid * 4);
        scale = glm::make_vec3(scales + gid * 3);
        quat_scale_to_covar_preci(quat, scale, &covar, nullptr);
    }
    glm::vec3 mean_c;
    pos_world_to_cam(R, t, glm::make_vec3(means), mean_c);
    glm::mat3 covar_c;
    covar_world_to_cam(R, covar, covar_c);

    // vjp: perspective projection
    float fx = Ks[0], cx = Ks[2], fy = Ks[4], cy = Ks[5];
    glm::mat3 v_covar_c(0.f);
    glm::vec3 v_mean_c(0.f);
    persp_proj_vjp(mean_c, covar_c, fx, fy, cx, cy, image_width, image_height,
                   v_covar2d, glm::make_vec2(v_means2d), v_mean_c, v_covar_c);

    // add contribution from v_depths
    v_mean_c.z += v_depths[0];

    // vjp: transform Gaussian covariance to camera space
    glm::vec3 v_mean(0.f);
    glm::mat3 v_covar(0.f);
    glm::mat3 v_R(0.f);
    glm::vec3 v_t(0.f);
    pos_world_to_cam_vjp(R, t, glm::make_vec3(means), v_mean_c, v_R, v_t, v_mean);
    covar_world_to_cam_vjp(R, covar, v_covar_c, v_R, v_covar);

    // #if __CUDA_ARCH__ >= 700
    // write out results with warp-level reduction
    auto warp = cg::tiled_partition<32>(cg::this_thread_block());
    auto warp_group_g = cg::labeled_partition(warp, gid);
    if (v_means != nullptr) {
        warpSum(v_mean, warp_group_g);
        if (warp_group_g.thread_rank() == 0) {
            v_means += gid * 3;
            PRAGMA_UNROLL
            for (uint32_t i = 0; i < 3; i++) {
                atomicAdd(v_means + i, v_mean[i]);
            }
        }
    }
    if (v_covars != nullptr) {
        // Output gradients w.r.t. the covariance matrix
        warpSum(v_covar, warp_group_g);
        if (warp_group_g.thread_rank() == 0) {
            v_covars += gid * 6;
            atomicAdd(v_covars, v_covar[0][0]);
            atomicAdd(v_covars + 1, v_covar[0][1] + v_covar[1][0]);
            atomicAdd(v_covars + 2, v_covar[0][2] + v_covar[2][0]);
            atomicAdd(v_covars + 3, v_covar[1][1]);
            atomicAdd(v_covars + 4, v_covar[1][2] + v_covar[2][1]);
            atomicAdd(v_covars + 5, v_covar[2][2]);
        }
    } else {
        // Directly output gradients w.r.t. the quaternion and scale
        glm::mat3 rotmat = quat_to_rotmat(quat);
        glm::vec4 v_quat(0.f);
        glm::vec3 v_scale(0.f);
        quat_scale_to_covar_vjp(quat, scale, rotmat, v_covar, v_quat, v_scale);
        warpSum(v_quat, warp_group_g);
        warpSum(v_scale, warp_group_g);
        if (warp_group_g.thread_rank() == 0) {
            v_quats += gid * 4;
            v_scales += gid * 3;
            atomicAdd(v_quats, v_quat[0]);
            atomicAdd(v_quats + 1, v_quat[1]);
            atomicAdd(v_quats + 2, v_quat[2]);
            atomicAdd(v_quats + 3, v_quat[3]);
            atomicAdd(v_scales, v_scale[0]);
            atomicAdd(v_scales + 1, v_scale[1]);
            atomicAdd(v_scales + 2, v_scale[2]);
        }
    }
    if (v_viewmats != nullptr) {
        auto warp_group_c = cg::labeled_partition(warp, cid);
        warpSum(v_R, warp_group_c);
        warpSum(v_t, warp_group_c);
        if (warp_group_c.thread_rank() == 0) {
            v_viewmats += cid * 16;
            PRAGMA_UNROLL
            for (uint32_t i = 0; i < 3; i++) { // rows
                PRAGMA_UNROLL
                for (uint32_t j = 0; j < 3; j++) { // cols
                    atomicAdd(v_viewmats + i * 4 + j, v_R[j][i]);
                }
                atomicAdd(v_viewmats + i * 4 + 3, v_t[i]);
            }
        }
    }
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
fully_fused_projection_fwd_tensor(
    const torch::Tensor &means,                // [N, 3]
    const at::optional<torch::Tensor> &covars, // [N, 6] optional
    const at::optional<torch::Tensor> &quats,  // [N, 4] optional
    const at::optional<torch::Tensor> &scales, // [N, 3] optional
    const torch::Tensor &viewmats,             // [C, 4, 4]
    const torch::Tensor &Ks,                   // [C, 3, 3]
    const uint32_t image_width, const uint32_t image_height, const float eps2d,
    const float near_plane, const float far_plane, const float radius_clip,
    const bool calc_compensations) {
    DEVICE_GUARD(means);
    CHECK_INPUT(means);
    if (covars.has_value()) {
        CHECK_INPUT(covars.value());
    } else {
        assert(quats.has_value() && scales.has_value());
        CHECK_INPUT(quats.value());
        CHECK_INPUT(scales.value());
    }
    CHECK_INPUT(viewmats);
    CHECK_INPUT(Ks);

    uint32_t N = means.size(0);    // number of gaussians
    uint32_t C = viewmats.size(0); // number of cameras
    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

    torch::Tensor radii = torch::empty({C, N}, means.options().dtype(torch::kInt32));
    torch::Tensor means2d = torch::empty({C, N, 2}, means.options());
    torch::Tensor depths = torch::empty({C, N}, means.options());
    torch::Tensor conics = torch::empty({C, N, 3}, means.options());
    torch::Tensor compensations;
    if (calc_compensations) {
        // we dont want NaN to appear in this tensor, so we zero intialize it
        compensations = torch::zeros({C, N}, means.options());
    }
    if (C && N) {
        fully_fused_projection_fwd_kernel<<<(C * N + N_THREADS - 1) / N_THREADS,
                                            N_THREADS, 0, stream>>>(
            C, N, means.data_ptr<float>(),
            covars.has_value() ? covars.value().data_ptr<float>() : nullptr,
            quats.has_value() ? quats.value().data_ptr<float>() : nullptr,
            scales.has_value() ? scales.value().data_ptr<float>() : nullptr,
            viewmats.data_ptr<float>(), Ks.data_ptr<float>(), image_width, image_height,
            eps2d, near_plane, far_plane, radius_clip, radii.data_ptr<int32_t>(),
            means2d.data_ptr<float>(), depths.data_ptr<float>(),
            conics.data_ptr<float>(),
            calc_compensations ? compensations.data_ptr<float>() : nullptr);
    }
    return std::make_tuple(radii, means2d, depths, conics, compensations);
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
fully_fused_projection_bwd_tensor(
    // fwd inputs
    const torch::Tensor &means,                // [N, 3]
    const at::optional<torch::Tensor> &covars, // [N, 6] optional
    const at::optional<torch::Tensor> &quats,  // [N, 4] optional
    const at::optional<torch::Tensor> &scales, // [N, 3] optional
    const torch::Tensor &viewmats,             // [C, 4, 4]
    const torch::Tensor &Ks,                   // [C, 3, 3]
    const uint32_t image_width, const uint32_t image_height, const float eps2d,
    // fwd outputs
    const torch::Tensor &radii,                       // [C, N]
    const torch::Tensor &conics,                      // [C, N, 3]
    const at::optional<torch::Tensor> &compensations, // [C, N] optional
    // grad outputs
    const torch::Tensor &v_means2d,                     // [C, N, 2]
    const torch::Tensor &v_depths,                      // [C, N]
    const torch::Tensor &v_conics,                      // [C, N, 3]
    const at::optional<torch::Tensor> &v_compensations, // [C, N] optional
    const bool viewmats_requires_grad) {
    DEVICE_GUARD(means);
    CHECK_INPUT(means);
    if (covars.has_value()) {
        CHECK_INPUT(covars.value());
    } else {
        assert(quats.has_value() && scales.has_value());
        CHECK_INPUT(quats.value());
        CHECK_INPUT(scales.value());
    }
    CHECK_INPUT(viewmats);
    CHECK_INPUT(Ks);
    CHECK_INPUT(radii);
    CHECK_INPUT(conics);
    CHECK_INPUT(v_means2d);
    CHECK_INPUT(v_depths);
    CHECK_INPUT(v_conics);
    if (compensations.has_value()) {
        CHECK_INPUT(compensations.value());
    }
    if (v_compensations.has_value()) {
        CHECK_INPUT(v_compensations.value());
        assert(compensations.has_value());
    }

    uint32_t N = means.size(0);    // number of gaussians
    uint32_t C = viewmats.size(0); // number of cameras
    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

    torch::Tensor v_means = torch::zeros_like(means);
    torch::Tensor v_covars, v_quats, v_scales; // optional
    if (covars.has_value()) {
        v_covars = torch::zeros_like(covars.value());
    } else {
        v_quats = torch::zeros_like(quats.value());
        v_scales = torch::zeros_like(scales.value());
    }
    torch::Tensor v_viewmats;
    if (viewmats_requires_grad) {
        v_viewmats = torch::zeros_like(viewmats);
    }
    if (C && N) {
        fully_fused_projection_bwd_kernel<<<(C * N + N_THREADS - 1) / N_THREADS,
                                            N_THREADS, 0, stream>>>(
            C, N, means.data_ptr<float>(),
            covars.has_value() ? covars.value().data_ptr<float>() : nullptr,
            covars.has_value() ? nullptr : quats.value().data_ptr<float>(),
            covars.has_value() ? nullptr : scales.value().data_ptr<float>(),
            viewmats.data_ptr<float>(), Ks.data_ptr<float>(), image_width, image_height,
            eps2d, radii.data_ptr<int32_t>(), conics.data_ptr<float>(),
            compensations.has_value() ? compensations.value().data_ptr<float>()
                                      : nullptr,
            v_means2d.data_ptr<float>(), v_depths.data_ptr<float>(),
            v_conics.data_ptr<float>(),
            v_compensations.has_value() ? v_compensations.value().data_ptr<float>()
                                        : nullptr,
            v_means.data_ptr<float>(),
            covars.has_value() ? v_covars.data_ptr<float>() : nullptr,
            covars.has_value() ? nullptr : v_quats.data_ptr<float>(),
            covars.has_value() ? nullptr : v_scales.data_ptr<float>(),
            viewmats_requires_grad ? v_viewmats.data_ptr<float>() : nullptr);
    }
    return std::make_tuple(v_means, v_covars, v_quats, v_scales, v_viewmats);
}

__global__ void fully_fused_projection_packed_fwd_kernel(
    const uint32_t C, const uint32_t N,
    const float *__restrict__ means,    // [N, 3]
    const float *__restrict__ covars,   // [N, 6] Optional
    const float *__restrict__ quats,    // [N, 4] Optional
    const float *__restrict__ scales,   // [N, 3] Optional
    const float *__restrict__ viewmats, // [C, 4, 4]
    const float *__restrict__ Ks,       // [C, 3, 3]
    const int32_t image_width, const int32_t image_height, const float eps2d,
    const float near_plane, const float far_plane, const float radius_clip,
    const int32_t *__restrict__ block_accum, // [C * blocks_per_row] packing helper
    int32_t *__restrict__ block_cnts,        // [C * blocks_per_row] packing helper
    // outputs
    int32_t *__restrict__ indptr,       // [C + 1]
    int64_t *__restrict__ camera_ids,   // [nnz]
    int64_t *__restrict__ gaussian_ids, // [nnz]
    int32_t *__restrict__ radii,        // [nnz]
    float *__restrict__ means2d,        // [nnz, 2]
    float *__restrict__ depths,         // [nnz]
    float *__restrict__ conics,         // [nnz, 3]
    float *__restrict__ compensations   // [nnz] optional
) {
    int32_t blocks_per_row = gridDim.x;

    int32_t row_idx = blockIdx.y; // cid
    int32_t block_col_idx = blockIdx.x;
    int32_t block_idx = row_idx * blocks_per_row + block_col_idx;

    int32_t col_idx = block_col_idx * blockDim.x + threadIdx.x; // gid

    bool valid = (row_idx < C) && (col_idx < N);

    // check if points are with camera near and far plane
    glm::vec3 mean_c;
    glm::mat3 R;
    if (valid) {
        // shift pointers to the current camera and gaussian
        means += col_idx * 3;
        viewmats += row_idx * 16;

        // glm is column-major but input is row-major
        R = glm::mat3(viewmats[0], viewmats[4], viewmats[8], // 1st column
                      viewmats[1], viewmats[5], viewmats[9], // 2nd column
                      viewmats[2], viewmats[6], viewmats[10] // 3rd column
        );
        glm::vec3 t = glm::vec3(viewmats[3], viewmats[7], viewmats[11]);

        // transform Gaussian center to camera space
        pos_world_to_cam(R, t, glm::make_vec3(means), mean_c);
        if (mean_c.z < near_plane || mean_c.z > far_plane) {
            valid = false;
        }
    }

    // check if the perspective projection is valid.
    glm::mat2 covar2d;
    glm::vec2 mean2d;
    glm::mat2 covar2d_inv;
    float compensation;
    float det;
    if (valid) {
        // transform Gaussian covariance to camera space
        glm::mat3 covar;
        if (covars != nullptr) {
            // if a precomputed covariance is provided
            covars += col_idx * 6;
            covar = glm::mat3(covars[0], covars[1], covars[2], // 1st column
                              covars[1], covars[3], covars[4], // 2nd column
                              covars[2], covars[4], covars[5]  // 3rd column
            );
        } else {
            // if not then compute it from quaternions and scales
            quats += col_idx * 4;
            scales += col_idx * 3;
            quat_scale_to_covar_preci(glm::make_vec4(quats), glm::make_vec3(scales),
                                      &covar, nullptr);
        }
        glm::mat3 covar_c;
        covar_world_to_cam(R, covar, covar_c);

        // perspective projection
        Ks += row_idx * 9;
        persp_proj(mean_c, covar_c, Ks[0], Ks[4], Ks[2], Ks[5], image_width,
                   image_height, covar2d, mean2d);

        det = add_blur(eps2d, covar2d, compensation);
        if (det <= 0.f) {
            valid = false;
        } else {
            // compute the inverse of the 2d covariance
            inverse(covar2d, covar2d_inv);
        }
    }

    // check if the points are in the image region
    float radius;
    if (valid) {
        // take 3 sigma as the radius (non differentiable)
        float b = 0.5f * (covar2d[0][0] + covar2d[1][1]);
        float v1 = b + sqrt(max(0.1f, b * b - det));
        float v2 = b - sqrt(max(0.1f, b * b - det));
        radius = ceil(3.f * sqrt(max(v1, v2)));

        if (radius <= radius_clip) {
            valid = false;
        }

        // mask out gaussians outside the image region
        if (mean2d.x + radius <= 0 || mean2d.x - radius >= image_width ||
            mean2d.y + radius <= 0 || mean2d.y - radius >= image_height) {
            valid = false;
        }
    }

    int32_t thread_data = static_cast<int32_t>(valid);
    if (block_cnts != nullptr) {
        // First pass: compute the block-wide sum
        int32_t aggregate;
        if (__syncthreads_or(thread_data)) {
            typedef hipcub::BlockReduce<int32_t, N_THREADS> BlockReduce;
            __shared__ typename BlockReduce::TempStorage temp_storage;
            aggregate = BlockReduce(temp_storage).Sum(thread_data);
        } else {
            aggregate = 0;
        }
        if (threadIdx.x == 0) {
            block_cnts[block_idx] = aggregate;
        }
    } else {
        // Second pass: write out the indices of the non zero elements
        if (__syncthreads_or(thread_data)) {
            typedef hipcub::BlockScan<int32_t, N_THREADS> BlockScan;
            __shared__ typename BlockScan::TempStorage temp_storage;
            BlockScan(temp_storage).ExclusiveSum(thread_data, thread_data);
        }
        if (valid) {
            if (block_idx > 0) {
                int32_t offset = block_accum[block_idx - 1];
                thread_data += offset;
            }
            // write to outputs
            camera_ids[thread_data] = row_idx;   // cid
            gaussian_ids[thread_data] = col_idx; // gid
            radii[thread_data] = (int32_t)radius;
            means2d[thread_data * 2] = mean2d.x;
            means2d[thread_data * 2 + 1] = mean2d.y;
            depths[thread_data] = mean_c.z;
            conics[thread_data * 3] = covar2d_inv[0][0];
            conics[thread_data * 3 + 1] = covar2d_inv[0][1];
            conics[thread_data * 3 + 2] = covar2d_inv[1][1];
            if (compensations != nullptr) {
                compensations[thread_data] = compensation;
            }
        }
        // lane 0 of the first block in each row writes the indptr
        if (threadIdx.x == 0 && block_col_idx == 0) {
            if (row_idx == 0) {
                indptr[0] = 0;
                indptr[C] = block_accum[C * blocks_per_row - 1];
            } else {
                indptr[row_idx] = block_accum[block_idx - 1];
            }
        }
    }
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor,
           torch::Tensor, torch::Tensor, torch::Tensor>
fully_fused_projection_packed_fwd_tensor(
    const torch::Tensor &means,                // [N, 3]
    const at::optional<torch::Tensor> &covars, // [N, 6]
    const at::optional<torch::Tensor> &quats,  // [N, 3]
    const at::optional<torch::Tensor> &scales, // [N, 3]
    const torch::Tensor &viewmats,             // [C, 4, 4]
    const torch::Tensor &Ks,                   // [C, 3, 3]
    const uint32_t image_width, const uint32_t image_height, const float eps2d,
    const float near_plane, const float far_plane, const float radius_clip,
    const bool calc_compensations) {
    DEVICE_GUARD(means);
    CHECK_INPUT(means);
    if (covars.has_value()) {
        CHECK_INPUT(covars.value());
    } else {
        assert(quats.has_value() && scales.has_value());
        CHECK_INPUT(quats.value());
        CHECK_INPUT(scales.value());
    }
    CHECK_INPUT(viewmats);
    CHECK_INPUT(Ks);

    uint32_t N = means.size(0);    // number of gaussians
    uint32_t C = viewmats.size(0); // number of cameras
    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
    auto opt = means.options().dtype(torch::kInt32);

    uint32_t nrows = C;
    uint32_t ncols = N;
    uint32_t blocks_per_row = (ncols + N_THREADS - 1) / N_THREADS;

    dim3 threads = {N_THREADS, 1, 1};
    // limit on the number of blocks: [2**31 - 1, 65535, 65535]
    dim3 blocks = {blocks_per_row, nrows, 1};

    // first pass
    int32_t nnz;
    torch::Tensor block_accum;
    if (C && N) {
        torch::Tensor block_cnts = torch::empty({nrows * blocks_per_row}, opt);
        fully_fused_projection_packed_fwd_kernel<<<blocks, threads, 0, stream>>>(
            C, N, means.data_ptr<float>(),
            covars.has_value() ? covars.value().data_ptr<float>() : nullptr,
            quats.has_value() ? quats.value().data_ptr<float>() : nullptr,
            scales.has_value() ? scales.value().data_ptr<float>() : nullptr,
            viewmats.data_ptr<float>(), Ks.data_ptr<float>(), image_width, image_height,
            eps2d, near_plane, far_plane, radius_clip, nullptr,
            block_cnts.data_ptr<int32_t>(), nullptr, nullptr, nullptr, nullptr, nullptr,
            nullptr, nullptr, nullptr);
        block_accum = torch::cumsum(block_cnts, 0, torch::kInt32);
        nnz = block_accum[-1].item<int32_t>();
    } else {
        nnz = 0;
    }

    // second pass
    torch::Tensor indptr = torch::empty({C + 1}, opt);
    torch::Tensor camera_ids = torch::empty({nnz}, opt.dtype(torch::kInt64));
    torch::Tensor gaussian_ids = torch::empty({nnz}, opt.dtype(torch::kInt64));
    torch::Tensor radii = torch::empty({nnz}, means.options().dtype(torch::kInt32));
    torch::Tensor means2d = torch::empty({nnz, 2}, means.options());
    torch::Tensor depths = torch::empty({nnz}, means.options());
    torch::Tensor conics = torch::empty({nnz, 3}, means.options());
    torch::Tensor compensations;
    if (calc_compensations) {
        // we dont want NaN to appear in this tensor, so we zero intialize it
        compensations = torch::zeros({nnz}, means.options());
    }

    if (nnz) {
        fully_fused_projection_packed_fwd_kernel<<<blocks, threads, 0, stream>>>(
            C, N, means.data_ptr<float>(),
            covars.has_value() ? covars.value().data_ptr<float>() : nullptr,
            quats.has_value() ? quats.value().data_ptr<float>() : nullptr,
            scales.has_value() ? scales.value().data_ptr<float>() : nullptr,
            viewmats.data_ptr<float>(), Ks.data_ptr<float>(), image_width, image_height,
            eps2d, near_plane, far_plane, radius_clip, block_accum.data_ptr<int32_t>(),
            nullptr, indptr.data_ptr<int32_t>(), camera_ids.data_ptr<int64_t>(),
            gaussian_ids.data_ptr<int64_t>(), radii.data_ptr<int32_t>(),
            means2d.data_ptr<float>(), depths.data_ptr<float>(),
            conics.data_ptr<float>(),
            calc_compensations ? compensations.data_ptr<float>() : nullptr);
    } else {
        indptr.fill_(0);
    }

    return std::make_tuple(indptr, camera_ids, gaussian_ids, radii, means2d, depths,
                           conics, compensations);
}

__global__ void fully_fused_projection_packed_bwd_kernel(
    // fwd inputs
    const uint32_t C, const uint32_t N, const uint32_t nnz,
    const float *__restrict__ means,    // [N, 3]
    const float *__restrict__ covars,   // [N, 6] Optional
    const float *__restrict__ quats,    // [N, 4] Optional
    const float *__restrict__ scales,   // [N, 3] Optional
    const float *__restrict__ viewmats, // [C, 4, 4]
    const float *__restrict__ Ks,       // [C, 3, 3]
    const int32_t image_width, const int32_t image_height, const float eps2d,
    // fwd outputs
    const int64_t *__restrict__ camera_ids,   // [nnz]
    const int64_t *__restrict__ gaussian_ids, // [nnz]
    const float *__restrict__ conics,         // [nnz, 3]
    const float *__restrict__ compensations,  // [nnz] optional
    // grad outputs
    const float *__restrict__ v_means2d,       // [nnz, 2]
    const float *__restrict__ v_depths,        // [nnz]
    const float *__restrict__ v_conics,        // [nnz, 3]
    const float *__restrict__ v_compensations, // [nnz] optional
    const bool sparse_grad, // whether the outputs are in COO format [nnz, ...]
    // grad inputs
    float *__restrict__ v_means,   // [N, 3] or [nnz, 3]
    float *__restrict__ v_covars,  // [N, 6] or [nnz, 6] Optional
    float *__restrict__ v_quats,   // [N, 4] or [nnz, 4] Optional
    float *__restrict__ v_scales,  // [N, 3] or [nnz, 3] Optional
    float *__restrict__ v_viewmats // [C, 4, 4] Optional
) {
    // parallelize over nnz.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= nnz) {
        return;
    }
    const int64_t cid = camera_ids[idx];   // camera id
    const int64_t gid = gaussian_ids[idx]; // gaussian id

    // shift pointers to the current camera and gaussian
    means += gid * 3;
    viewmats += cid * 16;
    Ks += cid * 9;

    conics += idx * 3;

    v_means2d += idx * 2;
    v_depths += idx;
    v_conics += idx * 3;

    // vjp: compute the inverse of the 2d covariance
    glm::mat2 covar2d_inv = glm::mat2(conics[0], conics[1], conics[1], conics[2]);
    glm::mat2 v_covar2d_inv =
        glm::mat2(v_conics[0], v_conics[1] * .5f, v_conics[1] * .5f, v_conics[2]);
    glm::mat2 v_covar2d(0.f);
    inverse_vjp(covar2d_inv, v_covar2d_inv, v_covar2d);

    if (v_compensations != nullptr) {
        // vjp: compensation term
        const float compensation = compensations[idx];
        const float v_compensation = v_compensations[idx];
        add_blur_vjp(eps2d, covar2d_inv, compensation, v_compensation, v_covar2d);
    }

    // transform Gaussian to camera space
    glm::mat3 R = glm::mat3(viewmats[0], viewmats[4], viewmats[8], // 1st column
                            viewmats[1], viewmats[5], viewmats[9], // 2nd column
                            viewmats[2], viewmats[6], viewmats[10] // 3rd column
    );
    glm::vec3 t = glm::vec3(viewmats[3], viewmats[7], viewmats[11]);
    glm::mat3 covar;
    glm::vec4 quat;
    glm::vec3 scale;
    if (covars != nullptr) {
        // if a precomputed covariance is provided
        covars += gid * 6;
        covar = glm::mat3(covars[0], covars[1], covars[2], // 1st column
                          covars[1], covars[3], covars[4], // 2nd column
                          covars[2], covars[4], covars[5]  // 3rd column
        );
    } else {
        // if not then compute it from quaternions and scales
        quat = glm::make_vec4(quats + gid * 4);
        scale = glm::make_vec3(scales + gid * 3);
        quat_scale_to_covar_preci(quat, scale, &covar, nullptr);
    }
    glm::vec3 mean_c;
    pos_world_to_cam(R, t, glm::make_vec3(means), mean_c);
    glm::mat3 covar_c;
    covar_world_to_cam(R, covar, covar_c);

    // vjp: perspective projection
    float fx = Ks[0], cx = Ks[2], fy = Ks[4], cy = Ks[5];
    glm::mat3 v_covar_c(0.f);
    glm::vec3 v_mean_c(0.f);
    persp_proj_vjp(mean_c, covar_c, fx, fy, cx, cy, image_width, image_height,
                   v_covar2d, glm::make_vec2(v_means2d), v_mean_c, v_covar_c);

    // add contribution from v_depths
    v_mean_c.z += v_depths[0];

    // vjp: transform Gaussian covariance to camera space
    glm::vec3 v_mean(0.f);
    glm::mat3 v_covar(0.f);
    glm::mat3 v_R(0.f);
    glm::vec3 v_t(0.f);
    pos_world_to_cam_vjp(R, t, glm::make_vec3(means), v_mean_c, v_R, v_t, v_mean);
    covar_world_to_cam_vjp(R, covar, v_covar_c, v_R, v_covar);

    auto warp = cg::tiled_partition<32>(cg::this_thread_block());
    if (sparse_grad) {
        // write out results with sparse layout
        if (v_means != nullptr) {
            v_means += idx * 3;
            PRAGMA_UNROLL
            for (uint32_t i = 0; i < 3; i++) {
                v_means[i] = v_mean[i];
            }
        }
        if (v_covars != nullptr) {
            v_covars += idx * 6;
            v_covars[0] = v_covar[0][0];
            v_covars[1] = v_covar[0][1] + v_covar[1][0];
            v_covars[2] = v_covar[0][2] + v_covar[2][0];
            v_covars[3] = v_covar[1][1];
            v_covars[4] = v_covar[1][2] + v_covar[2][1];
            v_covars[5] = v_covar[2][2];
        } else {
            glm::mat3 rotmat = quat_to_rotmat(quat);
            glm::vec4 v_quat(0.f);
            glm::vec3 v_scale(0.f);
            quat_scale_to_covar_vjp(quat, scale, rotmat, v_covar, v_quat, v_scale);
            v_quats += idx * 4;
            v_scales += idx * 3;
            v_quats[0] = v_quat[0];
            v_quats[1] = v_quat[1];
            v_quats[2] = v_quat[2];
            v_quats[3] = v_quat[3];
            v_scales[0] = v_scale[0];
            v_scales[1] = v_scale[1];
            v_scales[2] = v_scale[2];
        }
    } else {
        // write out results with dense layout
        // #if __CUDA_ARCH__ >= 700
        // write out results with warp-level reduction
        auto warp_group_g = cg::labeled_partition(warp, gid);
        if (v_means != nullptr) {
            warpSum(v_mean, warp_group_g);
            if (warp_group_g.thread_rank() == 0) {
                v_means += gid * 3;
                PRAGMA_UNROLL
                for (uint32_t i = 0; i < 3; i++) {
                    atomicAdd(v_means + i, v_mean[i]);
                }
            }
        }
        if (v_covars != nullptr) {
            // Directly output gradients w.r.t. the covariance
            warpSum(v_covar, warp_group_g);
            if (warp_group_g.thread_rank() == 0) {
                v_covars += gid * 6;
                atomicAdd(v_covars, v_covar[0][0]);
                atomicAdd(v_covars + 1, v_covar[0][1] + v_covar[1][0]);
                atomicAdd(v_covars + 2, v_covar[0][2] + v_covar[2][0]);
                atomicAdd(v_covars + 3, v_covar[1][1]);
                atomicAdd(v_covars + 4, v_covar[1][2] + v_covar[2][1]);
                atomicAdd(v_covars + 5, v_covar[2][2]);
            }
        } else {
            // Directly output gradients w.r.t. the quaternion and scale
            glm::mat3 rotmat = quat_to_rotmat(quat);
            glm::vec4 v_quat(0.f);
            glm::vec3 v_scale(0.f);
            quat_scale_to_covar_vjp(quat, scale, rotmat, v_covar, v_quat, v_scale);
            warpSum(v_quat, warp_group_g);
            warpSum(v_scale, warp_group_g);
            if (warp_group_g.thread_rank() == 0) {
                v_quats += gid * 4;
                v_scales += gid * 3;
                atomicAdd(v_quats, v_quat[0]);
                atomicAdd(v_quats + 1, v_quat[1]);
                atomicAdd(v_quats + 2, v_quat[2]);
                atomicAdd(v_quats + 3, v_quat[3]);
                atomicAdd(v_scales, v_scale[0]);
                atomicAdd(v_scales + 1, v_scale[1]);
                atomicAdd(v_scales + 2, v_scale[2]);
            }
        }
    }
    // v_viewmats is always in dense layout
    if (v_viewmats != nullptr) {
        auto warp_group_c = cg::labeled_partition(warp, cid);
        warpSum(v_R, warp_group_c);
        warpSum(v_t, warp_group_c);
        if (warp_group_c.thread_rank() == 0) {
            v_viewmats += cid * 16;
            PRAGMA_UNROLL
            for (uint32_t i = 0; i < 3; i++) { // rows
                PRAGMA_UNROLL
                for (uint32_t j = 0; j < 3; j++) { // cols
                    atomicAdd(v_viewmats + i * 4 + j, v_R[j][i]);
                }
                atomicAdd(v_viewmats + i * 4 + 3, v_t[i]);
            }
        }
    }
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
fully_fused_projection_packed_bwd_tensor(
    // fwd inputs
    const torch::Tensor &means,                // [N, 3]
    const at::optional<torch::Tensor> &covars, // [N, 6]
    const at::optional<torch::Tensor> &quats,  // [N, 4]
    const at::optional<torch::Tensor> &scales, // [N, 3]
    const torch::Tensor &viewmats,             // [C, 4, 4]
    const torch::Tensor &Ks,                   // [C, 3, 3]
    const uint32_t image_width, const uint32_t image_height, const float eps2d,
    // fwd outputs
    const torch::Tensor &camera_ids,                  // [nnz]
    const torch::Tensor &gaussian_ids,                // [nnz]
    const torch::Tensor &conics,                      // [nnz, 3]
    const at::optional<torch::Tensor> &compensations, // [nnz] optional
    // grad outputs
    const torch::Tensor &v_means2d,                     // [nnz, 2]
    const torch::Tensor &v_depths,                      // [nnz]
    const torch::Tensor &v_conics,                      // [nnz, 3]
    const at::optional<torch::Tensor> &v_compensations, // [nnz] optional
    const bool viewmats_requires_grad, const bool sparse_grad) {
    DEVICE_GUARD(means);
    CHECK_INPUT(means);
    if (covars.has_value()) {
        CHECK_INPUT(covars.value());
    } else {
        assert(quats.has_value() && scales.has_value());
        CHECK_INPUT(quats.value());
        CHECK_INPUT(scales.value());
    }
    CHECK_INPUT(viewmats);
    CHECK_INPUT(Ks);
    CHECK_INPUT(camera_ids);
    CHECK_INPUT(gaussian_ids);
    CHECK_INPUT(conics);
    CHECK_INPUT(v_means2d);
    CHECK_INPUT(v_depths);
    CHECK_INPUT(v_conics);
    if (compensations.has_value()) {
        CHECK_INPUT(compensations.value());
    }
    if (v_compensations.has_value()) {
        CHECK_INPUT(v_compensations.value());
        assert(compensations.has_value());
    }

    uint32_t N = means.size(0);    // number of gaussians
    uint32_t C = viewmats.size(0); // number of cameras
    uint32_t nnz = camera_ids.size(0);
    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

    torch::Tensor v_means, v_covars, v_quats, v_scales, v_viewmats;
    if (sparse_grad) {
        v_means = torch::zeros({nnz, 3}, means.options());
        if (covars.has_value()) {
            v_covars = torch::zeros({nnz, 6}, covars.value().options());
        } else {
            v_quats = torch::zeros({nnz, 4}, quats.value().options());
            v_scales = torch::zeros({nnz, 3}, scales.value().options());
        }
        if (viewmats_requires_grad) {
            v_viewmats = torch::zeros({C, 4, 4}, viewmats.options());
        }
    } else {
        v_means = torch::zeros_like(means);
        if (covars.has_value()) {
            v_covars = torch::zeros_like(covars.value());
        } else {
            v_quats = torch::zeros_like(quats.value());
            v_scales = torch::zeros_like(scales.value());
        }
        if (viewmats_requires_grad) {
            v_viewmats = torch::zeros_like(viewmats);
        }
    }
    if (nnz) {
        fully_fused_projection_packed_bwd_kernel<<<(nnz + N_THREADS - 1) / N_THREADS,
                                                   N_THREADS, 0, stream>>>(
            C, N, nnz, means.data_ptr<float>(),
            covars.has_value() ? covars.value().data_ptr<float>() : nullptr,
            covars.has_value() ? nullptr : quats.value().data_ptr<float>(),
            covars.has_value() ? nullptr : scales.value().data_ptr<float>(),
            viewmats.data_ptr<float>(), Ks.data_ptr<float>(), image_width, image_height,
            eps2d, camera_ids.data_ptr<int64_t>(), gaussian_ids.data_ptr<int64_t>(),
            conics.data_ptr<float>(),
            compensations.has_value() ? compensations.value().data_ptr<float>()
                                      : nullptr,
            v_means2d.data_ptr<float>(), v_depths.data_ptr<float>(),
            v_conics.data_ptr<float>(),
            v_compensations.has_value() ? v_compensations.value().data_ptr<float>()
                                        : nullptr,
            sparse_grad, v_means.data_ptr<float>(),
            covars.has_value() ? v_covars.data_ptr<float>() : nullptr,
            covars.has_value() ? nullptr : v_quats.data_ptr<float>(),
            covars.has_value() ? nullptr : v_scales.data_ptr<float>(),
            viewmats_requires_grad ? v_viewmats.data_ptr<float>() : nullptr);
    }
    return std::make_tuple(v_means, v_covars, v_quats, v_scales, v_viewmats);
}
